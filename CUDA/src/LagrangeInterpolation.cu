#include "hip/hip_runtime.h"
#include<stdio.h>
<<<<<<< HEAD
<<<<<<< HEAD
<<<<<<< HEAD
__global__ void interpolate(float * x, float * y, float a, float * k, int n){
    int i,j;
    i = blockIdx.x * blockDim.x + threadIdx.x;

    __shared__ float ss[2000], ts[2000], ks[2000];
    
    if(i<n)
    {
        ss[i]=1;
        ts[i]=1;
        __syncthreads();

        for(j=0;j<n;j++)
        {
            if(j!=i)
            {
                ss[i]=ss[i]*(a-x[j]);
                ts[i]=ts[i]*(x[i]-x[j]);
            }
        }

        ks[i]=(((ss[i])/(ts[i]))*y[i]);
	__syncthreads();
	if(i==0){
	    for(i=0;i<(n*n);i++){
	        *k += ks[i];
	    }
	}
    }
}

int main()
{
    float *x, *y, *d_x, *d_y;
    float a, k, *d_k;  
    size_t size = 2000 * sizeof(float);
    hipEvent_t start, stop;
    
    x = new float[2000];
    y = new float[2000];
    hipMalloc(&d_x, size);
    hipMalloc(&d_y, size);
    hipMalloc(&d_k, sizeof(float));
    hipEventCreate(&start);
    hipEventCreate(&stop);


    int N,i,d=1;
    printf("\n\n Enter the number of the terms of the table: ");
    scanf("%d",&N);
    printf("%d", N);
    printf("\n\n Enter the respective values of the variables x and y: \n");
    for(i=0; i<N; i++)
=======
=======
>>>>>>> 60b9fc31328e83730f9afbaa5cae5f1b1c57f5a0
=======
>>>>>>> 60b9fc31328e83730f9afbaa5cae5f1b1c57f5a0
main()
{
    float x[100],y[100],a,s=1,t=1,k=0;
    int n,i,j,d=1;
    printf("\n\n Enter the number of the terms of the table: ");
    scanf("%d",&n);
    printf("\n\n Enter the respective values of the variables x and y: \n");
    for(i=0; i<n; i++)
<<<<<<< HEAD
<<<<<<< HEAD
>>>>>>> 60b9fc31328e83730f9afbaa5cae5f1b1c57f5a0
=======
>>>>>>> 60b9fc31328e83730f9afbaa5cae5f1b1c57f5a0
=======
>>>>>>> 60b9fc31328e83730f9afbaa5cae5f1b1c57f5a0
    {
        scanf ("%f",&x[i]);
        scanf("%f",&y[i]);
    }
    printf("\n\n The table you entered is as follows :\n\n");
<<<<<<< HEAD
<<<<<<< HEAD
<<<<<<< HEAD
    for(i=0; i<N; i++)
=======
    for(i=0; i<n; i++)
>>>>>>> 60b9fc31328e83730f9afbaa5cae5f1b1c57f5a0
=======
    for(i=0; i<n; i++)
>>>>>>> 60b9fc31328e83730f9afbaa5cae5f1b1c57f5a0
=======
    for(i=0; i<n; i++)
>>>>>>> 60b9fc31328e83730f9afbaa5cae5f1b1c57f5a0
    {
        printf("%0.3f\t%0.3f",x[i],y[i]);
        printf("\n");
    }
<<<<<<< HEAD
<<<<<<< HEAD
<<<<<<< HEAD

 
    
    hipMemcpy(d_x, x, size, hipMemcpyHostToDevice);
    hipMemcpy(d_y, y, size, hipMemcpyHostToDevice);
    //while(d==1)
    //{
        printf(" \n\n\n Enter the value of the x to find the respective value of y\n\n\n");
        scanf("%f",&a);
	printf("%f\n",a);

	int threads_per_block = 32; // A 16 x 16 block threads
  	int number_of_blocks = N/threads_per_block + 1;
	//dim3 threads_per_block(3, 3); // A 16 x 16 block threads
  	//dim3 number_of_blocks(N/threads_per_block.x + 1, N/threads_per_block.y + 1);
	hipEventRecord(start);
	interpolate<<<number_of_blocks,threads_per_block>>>(d_x, d_y, a, d_k, N);

        hipError_t err;
        err = hipGetLastError(); // `hipGetLastError` will return the error from above.
        if (err != hipSuccess)
        { 
            printf("Error: %s\n", hipGetErrorString(err));
        }
	hipEventSynchronize(stop);
  	float milliseconds = 0;
  	hipEventElapsedTime(&milliseconds, start, stop);

	hipMemcpy(&k, d_k, sizeof(float), hipMemcpyDeviceToHost);

        printf("\n\n The respective value of the variable y is: %f\n", k);
	printf(" Elapsed time in milliseconds: %f\n", milliseconds);
        printf("\n\n Do you want to continue?\n\n Press 1 to continue and any other key to exit");
        scanf("%d",&d);
    //}

    delete [] x;
    delete [] y;
    hipFree(d_x);
    hipFree(d_y);
=======
=======
>>>>>>> 60b9fc31328e83730f9afbaa5cae5f1b1c57f5a0
=======
>>>>>>> 60b9fc31328e83730f9afbaa5cae5f1b1c57f5a0
    while(d==1)
    {
        printf(" \n\n\n Enter the value of the x to find the respective value of y\n\n\n");
        scanf("%f",&a);
        for(i=0; i<n; i++)
        {
            s=1;
            t=1;
            for(j=0; j<n; j++)
            {
                if(j!=i)
                {
                    s=s*(a-x[j]);
                    t=t*(x[i]-x[j]);
                }
            }
            k=k+((s/t)*y[i]);
        }
        printf("\n\n The respective value of the variable y is: %f",k);
        printf("\n\n Do you want to continue?\n\n Press 1 to continue and any other key to exit");
        scanf("%d",&d);
    }
<<<<<<< HEAD
<<<<<<< HEAD
>>>>>>> 60b9fc31328e83730f9afbaa5cae5f1b1c57f5a0
=======
>>>>>>> 60b9fc31328e83730f9afbaa5cae5f1b1c57f5a0
=======
>>>>>>> 60b9fc31328e83730f9afbaa5cae5f1b1c57f5a0
}
